/* jegood Joshua Good */

/**
 * @file p3.cu
 * Calculates the minimum distance for a set of file-specified points using GPU 
 * multi-threading. This program requires access to a CUDA-enabled GPU (i.e. NVIDIA 
 * graphics card).
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <limits.h>
#include <math.h>
#include <time.h>

/** Maximum number of threads per block */
#define MAX_THRDS 1024

// Point struct
struct point{
int x;
int y;
int index;
double minDistance;
}typedef Point;

/**
 * Calculates the minimum distance for this point from each point
 * in the points array.
 * @param points the point array
 * @param numPoints number of points in the point array
 */
__global__ void calcMinDist(Point *points, int numPoints)
{
	// Compute the minimum distance for each point in the point array
	for(int i = 0; i < numPoints; i++){
		// Ensure we don't calculate the distance to a point from itself
		if(i != points[blockIdx.x].index){
			double distance = sqrt(pow((double)(points[i].x - points[blockIdx.x].x), 2) + pow((double)(points[i].y - points[blockIdx.x].y), 2));
			// Check if distance is a new minimum distance for this point
			if(distance < points[blockIdx.x].minDistance){
				points[blockIdx.x].minDistance = distance;
			}
		}
	}
}

/**
 * Calculates the minimum distance for a set of file-specified points using a CUDA
 * kernel function. Reports this information and its associated minimum distance points
 * alongside the time taken to complete this process.
 * @param argc number of command line arguments
 * @param argv list of command of line arguments
 */ 
int main(int argc, char *argv[])
{
	FILE *fp;
	// Ensure a valid file is given
	if(!(fp = fopen(argv[1], "r"))){
		printf("Usage: ./p3 <input file>\n");
		exit(EXIT_FAILURE);
	}
	
	/** Start time for a process */
	clock_t start;
	/** End time for a process */
	clock_t finish;
	// Start process clock
	start = clock();
	
	// Initially loop through and calculate the number of points in the file
	Point p;
	/** Number of points in the file */
	int numPoints = 0;
	while(fscanf(fp, "%d%d", &p.x, &p.y) == 2){ // read, but don't store(*)
		numPoints++;
	}
	
	// Rewind the file and assign points in the array of points
	rewind(fp);
	/** Index of point in points array */
	int index = 0;
	Point points[numPoints];
	for(int i = 0; i < numPoints; i++){
		// Scan in next point
		fscanf(fp, "%d %d", &p.x, &p.y);
		p.index = index;
		p.minDistance = INFINITY;
		points[i] = p;
		index++;
	}
	
	// Allocate memory for kernel threads
	double minDist = INFINITY;
	Point *arr_p;
	int size = numPoints * sizeof(Point);
	hipMalloc((void**)&arr_p, size);
	hipMemcpy(arr_p, points, size, hipMemcpyHostToDevice);
	
	// Launch the kernel to do work
	// Runs numPoints blocks with one thread each
	calcMinDist<<<numPoints, 1>>>(arr_p, numPoints);
	// Use result on host
	hipMemcpy(points, arr_p, size, hipMemcpyDeviceToHost);
	
	// Determine minDist for these points
	for(int i = 0; i < numPoints; i++){
		if(points[i].minDistance < minDist){
			minDist = points[i].minDistance;
		}
	}
	
	// Determine which points have minimum distance
	for(int i = 0; i < numPoints; i++){
		if(points[i].minDistance == minDist){
			printf("(%d,%d)", points[i].x, points[i].y);
		}
	}
	// Print the minimum distance for the set of points
	printf("%lf\n", minDist);
	
	// End process time
	finish = clock();
	// Print the process time
	printf("Time : %lf seconds\n", (double) (finish - start) / CLOCKS_PER_SEC);
	
	// Free memory
	hipFree(arr_p);
	
	// Return EXIT_SUCCESS
	return 0;
}
